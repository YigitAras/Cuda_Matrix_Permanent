
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string.h>
#include <sstream>
#include <omp.h>
#include <stdlib.h>
#include <math.h>

using namespace std;

__global__ void perm_kernel(double * d_XX,unsigned long long int CHUNKY,double *d_p,double *d_mat,unsigned long long int SIZE,int N,int THREADS){
    unsigned long long int ind = threadIdx.x + blockIdx.x * blockDim.x;
    extern __shared__ double SXX[];
    unsigned long long int start_loc = ind*CHUNKY;
    unsigned long long int start_of_XX = threadIdx.x*N;
      for(int j=0;j<N;j++){
        SXX[threadIdx.x*N+j] = d_XX[j];
      }
      double* my_XX = SXX + start_of_XX;
    __syncthreads();
    for(int i=0;i<N;i++){
      // since all arrays are flattened
      SXX[start_of_XX + i] = d_mat[N*(N-1)+ i];
      for(int j=0; j<N; j++){
        SXX[start_of_XX + i] -= ((double)d_mat[(j*N)+i]/2);
      }
    }

    unsigned long long int y = (start_loc>>1) ^ start_loc;
    for( int i=0;i<N;i++){
      for( int k=0;k<N;k++){
        if( ( (y >> k ) & 1LL ) == 1){
          SXX[start_of_XX+i] += d_mat[N*k+i];  //    M[i][k]
        }
      }
    }

    start_loc += 1;
    // GOTTO start from start_loc+1 then go until CHUNKY+1
    unsigned long long int LIMITER = start_loc+CHUNKY;
    // carefull last chunk start_loc might go one over...
    int ps = (start_loc & 1LL) == 0 ? -1:1;


    // do the calculations for the whole sha-bang
    double local_p = 0.0; // this for local, then reduce it to outer
    for(unsigned long long i = start_loc; (i < LIMITER) && (i < SIZE) ;i++){
        unsigned long long int y = (i>>1LL) ^ i; // gray code
        unsigned long long int yy = ( (i-1)>>1LL ) ^ (i-1); // i-1's gray-code
        unsigned long long int z = __ffsll( y ^ yy )-1;  // get the changing bit
        long long int s = ((y >> z)  & 1LL) == 1 ?  1:-1; // find changing bit

        double dd = 1.0;
        for(int j=0;j<N;j++){
          my_XX[j] += s * d_mat[N*z+j]; // M[j][Z]
          dd *= my_XX[j];
        }

        ps *= -1;
        local_p += ps * dd;
    }
    // do a reduction on the d_p !!!!
    atomicAdd(d_p,local_p);
}


void usage()
{
  cout << "USAGE: ./exec <filename> <machine no>" << endl;
  exit(0);
}

int main(int argc, const char** argv)
{

  if(argc != 3)
    usage();

  string line;

  const char* filename = argv[1];
  int MACHINE_NO = atoi(argv[2]);
  ifstream input (filename);
  if(input.fail())
    return 0;


  int N;
  int **M;
  getline(input,line);
  N = atoi(line.c_str());
  M = new int*[N];
  for(int i = 0; i < N; i ++){
    M[i] = new int[N];
  }


  int linectr = 0;
  while(getline(input,line)){
    stringstream ss(line);
    int temp;
    int ctr = 0;
    while(ss >> temp)
      M[linectr][ctr++] = temp;

    linectr++;
  }

// NEED TO FLATTEN THE ARRAY
double * data_as_array = new double[N*N]();

for(int i=0; i<N; i++){
  for(int j=0; j<N; j++){
    // colum-major order
    data_as_array[i*N + j] = (double)M[j][i];
  }
}

hipSetDevice(MACHINE_NO);
hipDeviceProp_t prop;
hipGetDeviceProperties(&prop, 0);
unsigned int sharedmem = prop.sharedMemPerBlock;

cout << "Here are the specs\n";
cout << "Shared mem per block: " << sharedmem << "\n";

int size_mat = N*N * sizeof(double);
double *d_mat;
double *d_p;
double *d_XX;
double p = 1.0;
double *x = (double*)malloc(sizeof(double)*N);

for(int i=0;i<N;i++){
  x[i]= M[i][N-1];
  for(int j=0;j<N;j++){
    x[i] -= ((double)M[i][j]/2);
  }
  p *= x[i];
}
unsigned long long int SIZE = (unsigned long long int)1 << (N-1);
unsigned long long int THREADS = 128;
unsigned long long int BLOCKS = 256;
if(N<5){
  THREADS = 16;
  BLOCKS = 1;
}else if(N<10){
  THREADS = 512;
  BLOCKS  =  4;
}else if(N<20){
  THREADS = 256;
  BLOCKS = 128;
}else if(N <= 45){
  THREADS = 128;
  BLOCKS = 256;
}

/*
while(SIZE < (THREADS*BLOCKS)){
  if(BLOCKS != 1){
    BLOCKS /=2;
  }else{
    THREADS /=2;
  }
}
*/
unsigned long long int CHUNKY = SIZE/(THREADS*BLOCKS);
cout << "Current thread to block;\n\t THREAD:  " << THREADS
                                << "\n\t BLOCKS:  " << BLOCKS << "\n";
double *x_s = (double*) malloc(sizeof(double)*N);
/*
cout<<"Here is the initial X array: \n";
for(int i=0;i<N;i++){
  printf("%.2lf ",x[i]);
}
printf("\n");
*/
/*
int tid = 2;
long long int sloc = tid*CHUNKY;
int yyy = (sloc >>1)^sloc;

for(int i=0;i<N;i++){
    x_s[i] = x[i];
  for(int k=0;k<N;k++){
    if(((yyy>>k) & 1) == 1) {x_s[i] += M[i][k];}
  }
}

long long int starts = tid*CHUNKY+1;
double local_p = 1.0;

for(int i=starts;i<starts+CHUNKY+1;i++){
  int y = (i>>1) ^ i;
  int yy = ( (i-1)>> 1) ^ (i-1);
  int z = __builtin_ctz(y^yy);
  int s = ((y>>z)& 1) == 1 ? 1:-1;
  int prodsign = (i & 1) == 0 ? 1:-1;
  double dd = 1.0;

  for(int j=0;j<N;j++){
    x_s[j] += (double)(s*M[j][z]);
    dd *= x_s[j];
  }
  local_p += (double)(prodsign*dd);
}
printf("Here is the p in CPU: %.2lf\n",local_p);
*/

/*
cout << "Here is the X array for " << CHUNKY << " \n";
for(int i=0;i<N;i++){
  printf("%.2lf ",x_s[i]);
}
printf("\n");
*/

double *XX = (double* )malloc(sizeof(double)*N*THREADS*BLOCKS);
memset(XX,0.0,sizeof(double)*N*THREADS*BLOCKS);
// memory moving magiac
cout << "Chunky is this: " << CHUNKY << " \n";
cout << "Size is this: " << SIZE << " \n";
cout << "N is this: " << N << " \n";


hipMalloc((void **)&d_XX,THREADS*BLOCKS*N*sizeof(double));
hipMalloc((void **)&d_mat,size_mat);
hipMalloc((void **)&d_p,sizeof(double));
cout << "Memory Allocated...\n";
hipMemcpy(d_XX,XX,THREADS*BLOCKS*N*sizeof(double),hipMemcpyHostToDevice);
hipMemcpy(d_mat,data_as_array,size_mat,hipMemcpyHostToDevice);
hipMemcpy(d_p,&p,sizeof(double),hipMemcpyHostToDevice);
hipDeviceSynchronize();
cout << "Memory Copied...\n";

// preprocess the fuck out of it
//preprocess<<<BLOCKS,THREADS>>>(d_XX,d_mat,SIZE,N,CHUNKY);

cout << "Algo starts now.. Hold on to your seats\n";
double start,end;
start = omp_get_wtime();

 perm_kernel<<<BLOCKS,THREADS,(sizeof(double)*THREADS*N)>>>(d_XX,CHUNKY,d_p,d_mat,SIZE,N,THREADS);

cout << "Kernel finished running...\n";
hipMemcpy(&p,d_p,sizeof(double),hipMemcpyDeviceToHost);
end = omp_get_wtime();
cout << "Memory re-copied from the device to host...\n";
 p*= (4*(N & 1) - 2);
cout << "Result is: " << p <<" \n";
double result = end-start;
cout <<"The time the kernel took: " << result << " ...\n";


hipFree(d_XX);
hipFree(d_mat);
  return 0;
}
