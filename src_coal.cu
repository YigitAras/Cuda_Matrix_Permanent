
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string.h>
#include <sstream>
#include <omp.h>
#include <stdlib.h>
#include <math.h>

using namespace std;
__device__ int getGlobIdx_1D_2D(){
    return blockIdx.x*blockDim.x*blockDim.y
                     +threadIdx.y * blockDim.x + threadIdx.x;
}
__global__ void preprocess(double * d_XX,double *d_mat,unsigned long long int SIZE,int N,unsigned long long int CHUNKY){
    unsigned long long int ind = threadIdx.x + blockIdx.x * blockDim.x;
    //unsigned long long int ind = getGlobIdx_1D_2D();
    unsigned long long int start_loc = ind*CHUNKY;
    unsigned long long int start_of_XX = ind*N;
    for(int i=0;i<N;i++){
      // since all arrays are flattened
      d_XX[start_of_XX + i] = d_mat[N*(N-1)+ i];
      for(int j=0; j<N; j++){
        d_XX[start_of_XX + i] -= ((double)d_mat[(j*N)+i]/2);
      }
    }

    unsigned long long int y = (start_loc>>1) ^ start_loc;
    for(int i=0;i<N;i++){
      for(int k=0;k<N;k++){
        if( ( (y >> k ) & 1LL ) == 1){
          d_XX[start_of_XX+i] += d_mat[N*k+i];  //    M[i][k]
      }
    }
  }
}



__global__ void perm_kernel(double * d_XX,unsigned long long int CHUNKY,double *d_p,double *d_mat,unsigned long long int SIZE,int N,unsigned long long int THREADS,unsigned long long int BLOCKS){
  unsigned long long int ind = threadIdx.x + blockIdx.x * blockDim.x;
  //unsigned long long int ind = getGlobIdx_1D_2D();
  unsigned long long int start_loc = ind*CHUNKY + 1;
  // GOTTO start from start_loc+1 then go until CHUNKY+1
  unsigned long long int LIMITER = start_loc+CHUNKY;
  // carefull last chunk start_loc might go one over...
  int ps = (start_loc & 1LL) == 0 ? -1:1;

  // do the calculations for the whole sha-bang
  double local_p = 0.0; // this for local, then reduce it to outer
  for(unsigned long long i = start_loc; (i < LIMITER) && (i < SIZE) ;i++){
      unsigned long long int y = (i>>1LL) ^ i; // gray code
      unsigned long long int yy = ( (i-1)>>1LL ) ^ (i-1); // i-1's gray-code
      long long int z = __ffsll( y ^ yy )-1;  // get the changing bit
      long long int s = ((y >> z)  & 1LL) == 1 ?  1:-1; // find changing bit

      double dd = 1.0;
      for(int j=0;j<N;j++){
        d_XX[(j*THREADS*BLOCKS)+ind] += s * d_mat[N*z+j]; // M[j][Z]
        dd *= d_XX[(j*THREADS*BLOCKS)+ind];
      }

      ps *= -1;
      local_p += ps * dd;
  }
  // do a reduction on the d_p !!!!
  atomicAdd(d_p,local_p);
}


void usage()
{
  cout << "USAGE: ./exec <filename> <machine no>" << endl;
  exit(0);
}

int main(int argc, const char** argv)
{

  if(argc != 3)
    usage();

  string line;

  const char* filename = argv[1];
  int MACHINE_NO = atoi(argv[2]);
  ifstream input (filename);
  if(input.fail())
    return 0;


  int N;
  int **M;
  getline(input,line);
  N = atoi(line.c_str());
  M = new int*[N];
  for(int i = 0; i < N; i ++){
    M[i] = new int[N];
  }


  int linectr = 0;
  while(getline(input,line)){
    stringstream ss(line);
    int temp;
    int ctr = 0;
    while(ss >> temp)
      M[linectr][ctr++] = temp;

    linectr++;
  }
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  unsigned int sharedmem = prop.sharedMemPerBlock;

  cout << "Here are the specs\n";
  cout << "Shared mem per block: " << sharedmem << "\n";
// NEED TO FLATTEN THE ARRAY
double * data_as_array = new double[N*N]();

for(int i=0; i<N; i++){
  for(int j=0; j<N; j++){
    // colum-major order
    data_as_array[i*N + j] = (double)M[j][i];
  }
}

hipSetDevice(MACHINE_NO);


int size_mat = N*N * sizeof(double);
double *d_mat;
double *d_p;
double *d_XX;
double p = 1.0;
double *x = (double*)malloc(sizeof(double)*N);

for(int i=0;i<N;i++){
  x[i]= M[i][N-1];
  for(int j=0;j<N;j++){
    x[i] -= ((double)M[i][j]/2);
  }
  p *= x[i];
}
unsigned long long int SIZE = (unsigned long long int)1 << (N-1);
unsigned long long int THREADS = 512;
unsigned long long int BLOCKS = 32*1024;



while(SIZE < (THREADS*BLOCKS)){
  if(BLOCKS != 1){
    BLOCKS /=2;
  }else{
    THREADS /=2;
  }
}


unsigned long long int CHUNKY = SIZE/(THREADS*BLOCKS);
cout << "Current thread to block;\n\t THREAD:  " << THREADS
                                << "\n\t BLOCKS:  " << BLOCKS << "\n";
double *x_s = (double*) malloc(sizeof(double)*N);
/*
cout<<"Here is the initial X array: \n";
for(int i=0;i<N;i++){
  printf("%.2lf ",x[i]);
}
printf("\n");
*/
/*
int tid = 2;
long long int sloc = tid*CHUNKY;
int yyy = (sloc >>1)^sloc;

for(int i=0;i<N;i++){
    x_s[i] = x[i];
  for(int k=0;k<N;k++){
    if(((yyy>>k) & 1) == 1) {x_s[i] += M[i][k];}
  }
}

long long int starts = tid*CHUNKY+1;
double local_p = 1.0;

for(int i=starts;i<starts+CHUNKY+1;i++){
  int y = (i>>1) ^ i;
  int yy = ( (i-1)>> 1) ^ (i-1);
  int z = __builtin_ctz(y^yy);
  int s = ((y>>z)& 1) == 1 ? 1:-1;
  int prodsign = (i & 1) == 0 ? 1:-1;
  double dd = 1.0;

  for(int j=0;j<N;j++){
    x_s[j] += (double)(s*M[j][z]);
    dd *= x_s[j];
  }
  local_p += (double)(prodsign*dd);
}
printf("Here is the p in CPU: %.2lf\n",local_p);
*/

/*
cout << "Here is the X array for " << CHUNKY << " \n";
for(int i=0;i<N;i++){
  printf("%.2lf ",x_s[i]);
}
printf("\n");
*/

double *XX = (double* )malloc(sizeof(double)*N*THREADS*BLOCKS);
memset(XX,0.0,sizeof(double)*N*THREADS*BLOCKS);
// memory moving magiac
cout << "Chunky is this: " << CHUNKY << " \n";
cout << "Size is this: " << SIZE << " \n";
cout << "N is this: " << N << " \n";


hipMalloc((void **)&d_XX,THREADS*BLOCKS*N*sizeof(double));
hipMalloc((void **)&d_mat,size_mat);
hipMalloc((void **)&d_p,sizeof(double));
cout << "Memory Allocated...\n";
hipMemcpy(d_XX,XX,THREADS*BLOCKS*N*sizeof(double),hipMemcpyHostToDevice);
hipMemcpy(d_mat,data_as_array,size_mat,hipMemcpyHostToDevice);
hipMemcpy(d_p,&p,sizeof(double),hipMemcpyHostToDevice);
hipDeviceSynchronize();
cout << "Memory Copied...\n";

// preprocess the fuck out of it
preprocess<<<BLOCKS,THREADS>>>(d_XX,d_mat,SIZE,N,CHUNKY);

hipMemcpy(XX,d_XX,THREADS*BLOCKS*N*sizeof(double),hipMemcpyDeviceToHost);
cout << "Preprocess finished running...\n";

double *XXX = (double*) malloc(sizeof(double)*N*THREADS*BLOCKS);
memset(XXX,0.0,sizeof(double)*N*THREADS*BLOCKS);

for(unsigned long long int i=0;i<N;i++){
  for(unsigned long long int j=0;j<THREADS*BLOCKS;j++){
    XXX[j+(i*THREADS*BLOCKS)] = XX[i+(j*N)];
  }
}
cout << "Black magic is finished...\n";
// get it deer
hipMemcpy(d_XX,XXX,sizeof(double)*N*THREADS*BLOCKS,hipMemcpyHostToDevice);
/*
cout <<  "Here is the initial XX array: \n";
for(int i=0;i<N;i++){
  printf("%.2lf ",XX[i]);
}
printf("\n");
*/
/*
cout << "Here is the XX array for " << CHUNKY << " \n";
for(int i=0;i<N;i++){
  printf("%.2lf ",XX[N+i]);
}
printf("\n");
*/
cout << "Algo starts now.. Hold on to your seats\n";
double start,end;
start = omp_get_wtime();

 perm_kernel<<<BLOCKS,THREADS>>>(d_XX,CHUNKY,d_p,d_mat,SIZE,N,THREADS,BLOCKS);

hipMemcpy(&p,d_p,sizeof(double),hipMemcpyDeviceToHost);
end = omp_get_wtime();
cout << "Kernel finished running...\n";
cout << "Memory re-copied from the device to host...\n";
 p*= (4*(N & 1) - 2);
cout << "Result is: " << p <<" \n";
double result = end-start;
cout <<"The time the kernel took: " << result << " ...\n";


hipFree(d_XX);
hipFree(d_mat);
  return 0;
}
